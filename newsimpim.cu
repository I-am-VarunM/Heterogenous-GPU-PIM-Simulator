#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include<thrust/copy.h>
#define NUM_STREAMS 4 // Has to be played around to find the optimal value
#define TILE_WIDTH 16
namespace pim
{
    
    const int Numberofcolumns  = 1024; // per crossbar
    const int Numberofrows = 4; //per crossbar
    const int Numberofcrossbar =2;
    const int bitspermemristor = 2;
    int leftcrossbar = Numberofcrossbar; //To check how many crossbars are not occupied
    thrust::device_vector<int> smem(Numberofcrossbar*Numberofcolumns); // This is the memory for the crossbar. The crossbar outputs are accumulated here where the shift and add takes place
    thrust::host_vector<int> h_inputmatrix(M*N, 0); // Host Input Matrix 
    thrust::device_vector<int> d_input(M,0); // Device Input
    thrust::device_vector<int> weightmemory(Numberofcrossbar*Numberofcolumns*Numberofrows,0);
    thrust::host_vector<int> weightmemhost(Numberofcrossbar*Numberofcolumns*Numberofrows,0);
    thrust::device_vector<float> d_smem(Numberofcrossbar*Numberofrows*Numberofcolumns,0);
    /*
    cols should basically be a total cap on the number of the columsn totally including all the crossbars
    xsize is the x dimension of each vector
    ysize is the number of columns of each matrix
    rows and cols should be multiples of crossbar dimensions
    Before sending the input matrix rotate the Input matrix by 90 degrees.
    */
    
    __global__ void vectormultiplication(int *smem, float *d_smem,int *vec, int  *mat, int *out, const int rows, const int cols, const int xsize, const int ysize, int row_number){
        int tid=threadIdx.x+blockIdx.x*blockDim.x;
            int sum=0;

        int sharedMemSize = (cols + 31) / 32; // Integer division with ceiling effect
        /*
        //Suppose you want to multiply v1 and v2 to M1 and M2 represent that in this format one proper vector with [v1] and Matrix to be represented as [M1 M2]
                                                                                                                   [v2]
        */                                                                                                         
        if(tid<cols && (i+(tid/ysize)*xsize) < rows){ 
            for(int i=0; i<rows; i++)
                sum += vec[i + (tid/ysize)*xsize]*mat[(i*cols)+tid];
            out[tid]= sum;
            smem[tid/32] |= out[tid]<<(30-(bitspermemristor*tid)); //Mimicking the shift and add
            __syncthreads();
        }
        if (threadIdx.x < sharedMemSize) { //Copying the multiplied values from crossbar to Main memory.
            d_smem[blockIdx.x * sharedMemSize + threadIdx.x + row_number* cols] += __int_as_float(smem[threadIdx.x]); // accordingly some converting function has to be used here __int_as_float converts to IEEE FP32
        }
    }
    void cleardevicememory(float *d_smem) //clears device memory
    {
        thrust::fill(d_smem.begin(), d_smem.end(), 0);
    }
    void clearcrossbarmemory(int *smem) //Clears the crossbar outputs needs to be done everytime new vector matrix multiplication is done
    {
        thrust::fill(smem.begin(), smem.end(), 0);
    }
    /*sending the input matrix to multiply
    if the weight matrix is already loaded one function and if the weight matrix is not loaded one function
    M and N are the dimensions of Matrix
    */
    void matrix_multiplication(int *h_inputmatrix, int M, int N, int rows, int cols,int xsize, int ysize)
    {  hipStream_t streams[NUM_STREAMS];
        for (int i = 0; i < NUM_STREAMS; ++i) { hipStreamCreate(&streams[i]); }

        clearcrossbarmemory(thrust::raw_pointer_cast(smem.data()));
        
        for(int i=0;i<N;i++)
        {
            const int index= i%NUM_STREAMS;
            //Copying the host input to GPU memory
            hipMemcpyAsync(thrust::raw_pointer_cast(d_input.data()), thrust::raw_pointer_cast(h_inputmatrix.data())+i*M, d_input.size()*sizeof(mytype), hipMemcpyHostToDevice, streams[index]);
            //launching the kernel
            vectormultiplication<<<activeCrossbars,Numberofcolumns,Numberofcrossbar*Numberofrows*Numberofcolumns*sizeof(float), streams[i]>>>(
             thrust::raw_pointer_cast(smem.data()),
             thrust::raw_pointer_cast(d_smem.data()),
             thrust::raw_pointer_cast(d_input.data()),
             thrust::raw_pointer_cast(weightmemory.data()),
            rows, cols,
            xsize, ysize,
            i
         );
         clearcrossbarmemory(thrust::raw_pointer_cast(smem.data())); // Clearing the crossbar outputs
        }
        for (int i = 0; i < NUM_STREAMS; ++i) {
            hipStreamSynchronize(streams[i]);
        }
    
        for (int i = 0; i < NUM_STREAMS; ++i) {
            hipStreamDestroy(streams[i]);
        } 
    }

    void loadweightstopim(int *weightmatrix, int* weightmemory, int rows, int cols, int crossbarnumber)
    {
        thrust::copy(weightmatrix, weightmatrix + (rows * cols), weightmemory.begin() + crossbarnumber * (rows*cols));
    }
    void copyfromdevice(int *d_smem, int *h_smem,int rows, int cols, int start_crossbar, int end_crossbar)
    {
        thrust::copy(d_smem.begin()+(start_crossbar*rows*cols), d_smem.begin()+(end_crossbar*rows*cols), h_smem.begin());
    }
    void clearweights(int *weightmemory, int start_crossbar, int end_crossbar, int rows, int cols)
    {
        thrust::fill(weightmemory.begin() + (start_crossbar*rows*cols), weightmemory.begin() + (end_crossbar*rows*cols), 0); 
    }
    __global__ void softmax(float *kqtranspose, float *softmaxout, int rows, int cols)
    {
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        // Find the maximum value in the row
        float max_val = kqtranspose[row * cols + 0];
        for (int i = 1; i < cols; i++) {
            max_val = fmax(max_val, kqtranspose[row * cols + i]);
        }

        // Compute the numerator and denominator for Softmax
        float numerator = exp(kqtranspose[row * cols + col] - max_val);
        float denominator = 0.0f;
        for (int i = 0; i < cols; i++) {
            denominator += exp(kqtranspose[row * cols + i] - max_val);
        }

        // Compute the Softmax value
        softmaxout[row * cols + col] = numerator / denominator;

    } }
    void padmatrixandload(int *xbar, int *matrix, int M, int N, int Numberofcrossbar, int Numberofcolumns, int Numberofrows)
    {/*
        Explaining the function with an illustration
        suppose my matrix is 2x2 matrix but my crossbar size is 3x3
        [[1,2],[3,4]]
        This function is going to convert the matrix to [[1,2,0],[3,4,0],[0,0,0]]
        This fucntion is to maintain the uniformity of the crossbar
     */
     int x = (N+Numberofcolumns-1)/Numberofcolumns;
     int y = (M+Numberofrows-1)/Numberofrows;
     int reqxbar = x*y; //Required Number of crossbars for storing one matrix
     if(leftcrossbar > reqxbar)
     {
     for(int i=0;i<M;i++)
     {
        for(int j=0;j<N;j++)
        {
            xbar[(i)*Numberofcolumns + (j) + (Numberofcrossbar-leftcrossbar)*(Numberofcolumns*Numberofrows)] = matrix[i*N + j];
        }
     }
     //start = weightmemory.end(); //start index for loading the matrix
     //thrust::copy(xbarforparticularmatrix.begin(),xbarforparticularmatrix.end(), weightmemory.begin()+start);
     leftcrossbar -= reqxbar;
    }
}
//Matrix Multiplication (Tile Based considering the fact that Matrix sizes can be bigger than the maximum number of threads)
//GPU being used as GPU to perform matrix multiplication
__global__ void matrixMultiplyShared(float* A, float* B, float* C, int N, int M) {
    __shared__ float As[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Cvalue = 0.0;

    for (int t = 0; t < (M + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {
        if (Row < N && t * TILE_WIDTH + tx < M)
            As[ty][tx] = A[Row * M + t * TILE_WIDTH + tx];
        else
            As[ty][tx] = 0.0;

        if (Col < N && t * TILE_WIDTH + ty < M)
            Bs[ty][tx] = B[(t * TILE_WIDTH + ty) * N + Col];
        else
            Bs[ty][tx] = 0.0;

        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i) {
            Cvalue += As[ty][i] * Bs[i][tx];
        }

        __syncthreads();
    }

    if (Row < N && Col < N) {
        C[Row * N + Col] = Cvalue;
    }
}
    }