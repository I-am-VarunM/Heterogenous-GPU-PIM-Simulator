#include "driver.cuh"
//using namespace pim;
#include<iostream>
#include<vector>
#include<string>

namespace pim
{
void printNewRegMemory(int start_idx, int end_idx)
    {
        thrust::host_vector<float> h_smem(Numberofcrossbar*Numberofrows*Numberofcolumns);
        h_smem = gpu_mem;
        for(int i = start_idx; i < end_idx;i++)
           printf("%0.2f ",h_smem[i]);
        printf("\n------------\n");
    }

void printDevMemory(int start_idx, int end_idx)
    {
        thrust::host_vector<int> h_smem(Numberofcrossbar*Numberofrows*Numberofcolumns);
        h_smem = d_smem;
        for(int i = start_idx; i < end_idx;i++)
           printf("%d ",h_smem[i]);
        printf("\n------------\n");
    }
}
int main(int argc, char **argv)
{
	
	int M = 2,N = 2,total_size = 600;
        int* host = (int*)malloc(sizeof(int)*N*M);
        for(int i = 0;i < M*N;i++)
            host[i] = 1;
        int Xstart = 0, Xend = 1;

        int* weightmemory = (int*)malloc(sizeof(int)*total_size);
        for(int i = 0;i < total_size;i++)
            weightmemory[i] = 2;

        int* h_vector = (int*)malloc(sizeof(int)*M*N);
       // pim::loadweightstopim(weightmemory,0,total_size);
        pim::matrix_multiplication(host,M,N,Xstart,Xend,'G',0);
//        pim::copyfromXbartoGPU(0,1,0);
  //      pim::printNewRegMemory(0,256);
  //      printf("\n---------\n");
/*        for(int i = 0;i < M*N;i++)
            host[i] = 2;
        pim::matrix_multiplication(host,M,N,Xstart,Xend,'G',256);
        //pim::copyfromXbartoGPU(0,1,);
        pim::GPUmatmult(0,256,512,2,128);
        pim::printDevMemory(0,256);
        pim::scalardivDr(520,540,4,4,4);
        //pim::softmaxDr(530,550,4,4);
        pim::printNewRegMemory(520,570);
        //pim::GPUtoHost(40,50,h_vector);*/
        return 0;
}

